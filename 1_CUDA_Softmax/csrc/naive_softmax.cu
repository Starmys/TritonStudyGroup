#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>
#include <torch/extension.h>
#include <float.h>
#include <hip/hip_runtime.h>


__global__ void naive_softmax_kernel(float* x, float* y, int batch_size, int hidden_dim) {
    int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (row_idx >= batch_size) return;

    float max_val = -FLT_MAX;
    for (int i = 0; i < hidden_dim; i++) {
        max_val = max(max_val, x[row_idx * hidden_dim + i]);
    }

    float sum_exp = 0.0f;
    for (int i = 0; i < hidden_dim; i++) {
        float tmp_val = x[row_idx * hidden_dim + i];
        sum_exp += expf(tmp_val - max_val);
    }

    for (int i = 0; i < hidden_dim; i++) {
        float tmp_val = x[row_idx * hidden_dim + i];
        y[row_idx * hidden_dim + i] = expf(tmp_val - max_val) / sum_exp;
    }
}


torch::Tensor naive_softmax(torch::Tensor X) {
    hipSetDevice(X.get_device());

    int batch_size = X.size(0);
    int hidden_dim = X.size(1);
    torch::Tensor Y = torch::empty_like(X, X.options());

    const int num_threads = 128;
    int num_blocks = (batch_size + num_threads - 1) / num_threads;

    const dim3 dimBlock(num_threads);
    const dim3 dimGrid(num_blocks);
    naive_softmax_kernel<<<dimGrid, dimBlock>>>(
        X.data_ptr<float>(),
        Y.data_ptr<float>(),
        batch_size, hidden_dim
    );

    return Y;
}
